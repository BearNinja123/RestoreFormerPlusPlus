#include "hip/hip_runtime.h"
#include <ATen/native/SharedReduceOps.h> // WelfordData/WelfordOps
#include <ATen/native/cuda/Loops.cuh>
#include <c10/core/Device.h>
#include <c10/core/DeviceType.h>
#include <c10/core/ScalarType.h>
#include <c10/cuda/CUDAMathCompat.h> // rsqrt
#include <ATen/AccumulateType.h> // acc_type
#include <ATen/ATen.h>
#include <string>
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <thrust/pair.h>
#include <hip/hip_runtime.h>
#include <vector>
#define THREADS_PER_BLOCK 128 // low threads per block bad because less occupancy, high threads per block bad because of smaller reduction loops -> more instruction overhead

template <typename T>
__global__ void
compute_stats4(
        const T* X,
        const int D,
        const int G,
        const int HWC,
        const int num_elems_coalesced,
        const float eps,
        T* means,
        T* rstds
  ) {
  using T_ACC = at::acc_type<T, true>;
  using WelfordType = at::native::WelfordData<T_ACC, int>;
  using WelfordOp = at::native::WelfordOps<T_ACC, T_ACC, int, thrust::pair<T_ACC, T_ACC>>;
  // griddim = N, G/g, blockdim = num_elems_coalesced (gD), c

  WelfordOp welford_op = {/*correction=*/0, /*take_sqrt=*/false};
  WelfordType val(0, 0, 0, 0);

  __shared__ typename std::aligned_storage<sizeof(WelfordType), alignof(WelfordType)>::type vals_reduced_arr[THREADS_PER_BLOCK];
  WelfordType *vals_reduced = reinterpret_cast<WelfordType*>(vals_reduced_arr);

  const int HWd = HWC / gridDim.y / THREADS_PER_BLOCK;
#pragma unroll 8
  for (int i = 0; i < HWd; ++i) {
    int reduce_idx = i * blockDim.y * D * G + threadIdx.y * D * G + blockIdx.y * num_elems_coalesced + threadIdx.x; // only works if THREADS_PER_BLOCK >= D but realistically this will happen all the time
    T x = X[blockIdx.x * HWC + reduce_idx];
    val = welford_op.reduce(val, static_cast<T_ACC>(x), reduce_idx); // last arg isn't used in src
  }

  // suppose vals_reduced shape is (c, g, D), we need (g,) output
  // (c,g,D) -> (D,c,g) -> (g,) (where g * D = num_elems_coalesced)
  const int g = num_elems_coalesced / D; // number of groups this thread block is processing
  int tid = threadIdx.y * blockDim.x + threadIdx.x;
  const int c_idx = threadIdx.y;
  const int g_idx = threadIdx.x / D;
  const int d = threadIdx.x % D;
  vals_reduced[d * blockDim.y * g + c_idx * g + g_idx] = val;
  __syncthreads();

  for (int stride = THREADS_PER_BLOCK / 2; stride >= g; stride >>= 1) {
    if (tid < stride)
      vals_reduced[tid] = welford_op.combine(vals_reduced[tid], vals_reduced[tid + stride]);
    __syncthreads();
    }

  // put reduced outputs into return buffers
  if ((int)threadIdx.x < g && threadIdx.y == 0) {
    T_ACC m1, m2;
    thrust::tie(m2, m1) = welford_op.project(vals_reduced[threadIdx.x]);
    //thrust::tie(m2, m1) = welford_op.project(vals_reduced[blockDim.y * D * threadIdx.x]);
    means[blockIdx.x * G + blockIdx.y * g + threadIdx.x] = m1;
    rstds[blockIdx.x * G + blockIdx.y * g + threadIdx.x] = c10::cuda::compat::rsqrt(m2 + static_cast<T_ACC>(eps));
  }
}

template <typename T>
__global__ void
compute_scale_biases4(
        T* means,  // (N, G)
        T* rstds,  // (N, G)
        const T* weight, // (C)
        const T* bias,   // (C)
        const int G,
        const int C,
        at::acc_type<T, true>* a,            // (N, C)
        at::acc_type<T, true>* b             // (N, C)
  ) {
  const int D = C / G;
  //const int g = blockIdx.y * blockDim.y + threadIdx.y;
  //const int c = g * D + threadIdx.x;
  const int c = threadIdx.x;
  const int g = c / D;
  const int nc = blockIdx.x * C + c;
  const int ng = blockIdx.x * G + g;
  const at::acc_type<T, true> a_nc = rstds[ng] * weight[c];
  a[nc] = a_nc;
  b[nc] = -means[ng] * a_nc + bias[c];
}

template <typename T>
void gn_nhwc_forward_kernel4(
    const torch::Tensor& X,
    const torch::Tensor& weight,
    const torch::Tensor& bias,
    const int G,
    T eps,
    torch::Tensor& Y,
    torch::Tensor& means,
    torch::Tensor& rstds) {
  using T_ACC = at::acc_type<T, true>;
  const T* X_data = X.const_data_ptr<T>();
  T* mean_data = means.mutable_data_ptr<T>();
  T* rstd_data = rstds.mutable_data_ptr<T>();
  const T* weight_data = weight.const_data_ptr<T>();
  const T* bias_data = bias.const_data_ptr<T>();

  const int N = X.size(0);
  const int H = X.size(1);
  const int W = X.size(2);
  const int C = X.size(3);
  const int D = C / G;
  const int num_elems_coalesced = 8; // reads 8 floats consecutively, will still cause uncoalesced reads for bf16 (2 bytes/float * 8 floats -> 16 bytes, 16 bytes < 32 bytes/coalesced read) but happens to work better than reading 16 floats because of larger grid size
  int blockDimX, blockDimY, gridDimY, gridDimZ;
  blockDimX = num_elems_coalesced;
  blockDimY = THREADS_PER_BLOCK / blockDimX;
  gridDimY = C / blockDimX;
  gridDimZ = 1;

  dim3 dimGrid(N, gridDimY, gridDimZ);
  dim3 dimBlock(blockDimX, blockDimY);

  const int HWC = H * W * C;
  compute_stats4<T><<<dimGrid, dimBlock>>>(
      X_data, D, G, HWC, num_elems_coalesced, eps,
      mean_data, rstd_data
  );

  const at::ScalarType kAccType =
      (X.scalar_type() == at::kHalf || X.scalar_type() == at::kBFloat16)
      ? at::kFloat
      : X.scalar_type();

  torch::Tensor a = torch::empty({N, C}, X.options().dtype(kAccType));
  torch::Tensor b = torch::empty({N, C}, X.options().dtype(kAccType));
  T_ACC* a_data = a.mutable_data_ptr<T_ACC>();
  T_ACC* b_data = b.mutable_data_ptr<T_ACC>();

  compute_scale_biases4<<<N, C>>>( // note: max(D, T) threads per block
      mean_data, rstd_data,
      weight_data, bias_data,
      G, C,
      a_data, b_data);

  at::TensorIterator iter = at::TensorIteratorConfig()
    .check_all_same_dtype(std::is_same<T, T_ACC>::value) // this line relaxes requirement that all inputs/outputs are same dtype if T isn't T_ACC 
    .resize_outputs(false)
    .add_owned_output(Y.view({N, H * W, G, D}))
    .add_owned_input(X.view({N, H * W, G, D}))
    .add_owned_input(a.view({N, 1, G, D}))
    .add_owned_input(b.view({N, 1, G, D}))
    .build();

  at::native::gpu_kernel(iter, [] GPU_LAMBDA(T x, T_ACC a, T_ACC b) -> T {
      return static_cast<T_ACC>(x) * a + b;
      });
  AT_CUDA_CHECK(hipGetLastError());
}

std::vector<torch::Tensor> gn_nhwc_cuda_forward4(
    const torch::Tensor& X,
    const torch::Tensor& weight,
    const torch::Tensor& bias,
    const int G,
    float eps) {
  const int N = X.size(0);

  torch::Tensor X_nhwc = X.permute({0, 2, 3, 1});
  torch::Tensor X_out = torch::empty_like(X_nhwc);
  torch::Tensor means = torch::empty({N, G}, weight.options());
  torch::Tensor rstds = torch::empty({N, G}, weight.options());

  AT_DISPATCH_FLOATING_TYPES_AND2(
    at::ScalarType::Half,
    at::ScalarType::BFloat16,
    X.scalar_type(),
    "group_norm_nhwc_forward", [&]() {
      gn_nhwc_forward_kernel4<scalar_t>(
          X_nhwc,
          weight,
          bias,
          G,
          eps,
          X_out,
          means,
          rstds
      );
  });
  return {X_out.permute({0, 3, 1, 2}), means, rstds};
}
