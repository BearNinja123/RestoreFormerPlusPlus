#include "hip/hip_runtime.h"
#include <ATen/native/SharedReduceOps.h> // WelfordData/WelfordOps
#include <ATen/native/cuda/Loops.cuh>
#include <c10/core/Device.h>
#include <c10/core/DeviceType.h>
#include <c10/core/ScalarType.h>
#include <c10/cuda/CUDAMathCompat.h> // rsqrt
#include <ATen/AccumulateType.h> // acc_type
#include <ATen/ATen.h>
#include <string>
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <thrust/pair.h>
#include <hip/hip_runtime.h>
#include <vector>
#define THREADS_PER_BLOCK 128 // 512 slightly faster (~3%) than 1024 because of higher theoretical occupancy -> higher mem throughput

// Reduces a value across the y-threads of a threadblock
template <typename T, class ReduceOp>
__device__ void
full_reduce(
    T val,
    const ReduceOp& op,
    T* output_buffer
    ) {
  int tid = threadIdx.y * blockDim.x + threadIdx.x;
  output_buffer[tid] = val;
  __syncthreads();

  for (int stride = (int)(blockDim.x * blockDim.y / 2); stride >= 1; stride >>= 1) {
    if (tid < stride)
      output_buffer[tid] = op.combine(output_buffer[tid], output_buffer[tid + stride]);
    __syncthreads();
    }
}
template <typename T>
__global__ void
compute_stats2(
        const T* X,
        const int D,
        const int G,
        const int HWd,
        const float eps,
        T* means,
        T* rstds
  ) {
  using T_ACC = at::acc_type<T, true>;
  using WelfordType = at::native::WelfordData<T_ACC, int>;
  using WelfordOp = at::native::WelfordOps<T_ACC, T_ACC, int, thrust::pair<T_ACC, T_ACC>>;
  // griddim = N, G, blockdim = d, D

  WelfordOp welford_op = {/*correction=*/0, /*take_sqrt=*/false};
  WelfordType val(0, 0, 0, 0);

  __shared__ typename std::aligned_storage<sizeof(WelfordType), alignof(WelfordType)>::type vals_reduced_arr[THREADS_PER_BLOCK];
  WelfordType *vals_reduced = reinterpret_cast<WelfordType*>(vals_reduced_arr);

  const int HWC = HWd * THREADS_PER_BLOCK * G;
//#pragma unroll 8
  //for (int i = 0; i < HWd/4; ++i) {
  for (int i = 0; i < HWd; ++i) {
    //int reduce_idx = i * THREADS_PER_BLOCK * G * 4 + threadIdx.y * D * G + blockIdx.y * D + threadIdx.x * 4; // only works if THREADS_PER_BLOCK >= D but realistically this will happen all the time
    //const float4 tmp = reinterpret_cast<const float4 *>(X + (blockIdx.x * HWC + reduce_idx))[0];
    //val = welford_op.reduce(val, static_cast<T_ACC>(tmp.x), reduce_idx); // last arg isn't used in src
    //val = welford_op.reduce(val, static_cast<T_ACC>(tmp.y), reduce_idx); // last arg isn't used in src
    //val = welford_op.reduce(val, static_cast<T_ACC>(tmp.z), reduce_idx); // last arg isn't used in src
    //val = welford_op.reduce(val, static_cast<T_ACC>(tmp.w), reduce_idx); // last arg isn't used in src

    int reduce_idx = i * THREADS_PER_BLOCK * G + threadIdx.y * D * G + blockIdx.y * D + threadIdx.x; // only works if THREADS_PER_BLOCK >= D but realistically this will happen all the time
    T x = X[blockIdx.x * HWC + reduce_idx];
    val = welford_op.reduce(val, static_cast<T_ACC>(x), reduce_idx); // last arg isn't used in src
  }

  full_reduce(val, welford_op, vals_reduced);
  if (threadIdx.x == 0 && threadIdx.y == 0) {
    T_ACC m1, m2;
    thrust::tie(m2, m1) = welford_op.project(vals_reduced[threadIdx.x]);
    means[blockIdx.x * G + blockIdx.y] = m1;
    rstds[blockIdx.x * G + blockIdx.y] = c10::cuda::compat::rsqrt(m2 + static_cast<T_ACC>(eps));
  }
}

template <typename T>
__global__ void
compute_scale_biases2(
        T* means,  // (N, G)
        T* rstds,  // (N, G)
        const T* weight, // (C)
        const T* bias,   // (C)
        const int G,
        const int C,
        at::acc_type<T, true>* a,            // (N, C)
        at::acc_type<T, true>* b             // (N, C)
  ) {
  const int D = C / G;
  //const int g = blockIdx.y * blockDim.y + threadIdx.y;
  //const int c = g * D + threadIdx.x;
  const int c = threadIdx.x;
  const int g = c / D;
  const int nc = blockIdx.x * C + c;
  const int ng = blockIdx.x * G + g;
  const at::acc_type<T, true> a_nc = rstds[ng] * weight[c];
  a[nc] = a_nc;
  b[nc] = -means[ng] * a_nc + bias[c];
}

template <typename T>
void gn_nhwc_forward_kernel2(
    const torch::Tensor& X,
    const torch::Tensor& weight,
    const torch::Tensor& bias,
    const int G,
    T eps,
    torch::Tensor& Y,
    torch::Tensor& means,
    torch::Tensor& rstds) {
  using T_ACC = at::acc_type<T, true>;
  const T* X_data = X.const_data_ptr<T>();
  T* mean_data = means.mutable_data_ptr<T>();
  T* rstd_data = rstds.mutable_data_ptr<T>();
  const T* weight_data = weight.const_data_ptr<T>();
  const T* bias_data = bias.const_data_ptr<T>();

  const int N = X.size(0);
  const int H = X.size(1);
  const int W = X.size(2);
  const int C = X.size(3);
  const int D = C / G;
  int blockDimX, blockDimY, gridDimY, gridDimZ;
  //blockDimX = D / 4;
  blockDimX = D;
  blockDimY = THREADS_PER_BLOCK / blockDimX;
  gridDimY = G;
  gridDimZ = 1;

  dim3 dimGrid(N, gridDimY, gridDimZ);
  dim3 dimBlock(blockDimX, blockDimY);

  const int HWd = H * W * D / THREADS_PER_BLOCK;
  compute_stats2<T><<<dimGrid, dimBlock>>>(
      X_data, D, G, HWd, eps,
      mean_data, rstd_data
  );

  const at::ScalarType kAccType =
      (X.scalar_type() == at::kHalf || X.scalar_type() == at::kBFloat16)
      ? at::kFloat
      : X.scalar_type();

  torch::Tensor a = torch::empty({N, C}, X.options().dtype(kAccType));
  torch::Tensor b = torch::empty({N, C}, X.options().dtype(kAccType));
  T_ACC* a_data = a.mutable_data_ptr<T_ACC>();
  T_ACC* b_data = b.mutable_data_ptr<T_ACC>();
  
  if (THREADS_PER_BLOCK >= C) {
    blockDimX = C;
    blockDimY = THREADS_PER_BLOCK / C;
    gridDimY = 1;
  }
  else {
    blockDimX = THREADS_PER_BLOCK;
    blockDimY = 1;
    gridDimY = C / THREADS_PER_BLOCK;
  }

  const int TPB = 128;
  gridDimY = C / TPB;
  gridDimY = gridDimY > 0 ? gridDimY : 1;
  blockDimY = TPB >= D ? TPB / D : 1;
  //compute_scale_biases2<<<dim3(N, gridDimY), dim3(D, blockDimY)>>>( // note: max(D, T) threads per block
  compute_scale_biases2<<<N, C>>>( // note: max(D, T) threads per block
      mean_data, rstd_data,
      weight_data, bias_data,
      G, C,
      a_data, b_data);

  at::TensorIterator iter = at::TensorIteratorConfig()
    .check_all_same_dtype(std::is_same<T, T_ACC>::value) // this line relaxes requirement that all inputs/outputs are same dtype if T isn't T_ACC 
    .resize_outputs(false)
    .add_owned_output(Y.view({N, H * W, G, D}))
    .add_owned_input(X.view({N, H * W, G, D}))
    .add_owned_input(a.view({N, 1, G, D}))
    .add_owned_input(b.view({N, 1, G, D}))
    .build();

  at::native::gpu_kernel(iter, [] GPU_LAMBDA(T x, T_ACC a, T_ACC b) -> T {
      return static_cast<T_ACC>(x) * a + b;
      });
  AT_CUDA_CHECK(hipGetLastError());
}

std::vector<torch::Tensor> gn_nhwc_cuda_forward2(
    const torch::Tensor& X,
    const torch::Tensor& weight,
    const torch::Tensor& bias,
    const int G,
    float eps) {
  const int N = X.size(0);

  torch::Tensor X_nhwc = X.permute({0, 2, 3, 1});
  torch::Tensor X_out = torch::empty_like(X_nhwc);
  torch::Tensor means = torch::empty({N, G}, weight.options());
  torch::Tensor rstds = torch::empty({N, G}, weight.options());

  AT_DISPATCH_FLOATING_TYPES_AND2(
    at::ScalarType::Half,
    at::ScalarType::BFloat16,
    X.scalar_type(),
    "group_norm_nhwc_forward", [&]() {
      gn_nhwc_forward_kernel2<scalar_t>(
          X_nhwc,
          weight,
          bias,
          G,
          eps,
          X_out,
          means,
          rstds
      );
  });
  return {X_out.permute({0, 3, 1, 2}), means, rstds};
}
